#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>

#define NUM_DATA 512

__global__ void vecAdd(int* _a, int* _b, int* _c){
	int tID = threadIdx.x;
	_c[tID] = _a[tID] + _b[tID];
}

int main(void){
	int *a, *b, *c;
	int *d_a, *d_b, *d_c;

	int memSize = sizeof(int)*NUM_DATA;

	printf("%d elements, memsize = %d bytes\n", NUM_DATA, memSize);
	
	a = new int[NUM_DATA]; memset(a, 0, memSize);
	b = new int[NUM_DATA]; memset(b, 0, memSize);
	c = new int[NUM_DATA]; memset(c, 0, memSize);
	
	for (int i = 0; i < NUM_DATA; i++){
		a[i] = rand() % 10;
		b[i] = rand() % 10;
	}

	hipMalloc(&d_a, memSize);
	hipMalloc(&d_b, memSize);
	hipMalloc(&d_c, memSize);
	
	//Data Transfer overhead 1
	clock_t st = clock();
	hipMemcpy(d_a, a, memSize, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, memSize, hipMemcpyHostToDevice);
	clock_t et = clock();
	printf("Data Transfer Overhead 1 - Host to Device : %ldms\n", et - st);

	//computation
	vecAdd<<<1, NUM_DATA>>>(d_a, d_b, d_c);
	
	//Data Transfer overhead 2
	st = clock();
	hipMemcpy(c, d_c, memSize, hipMemcpyDeviceToHost);
	et = clock();
	printf("Data Transfer Overhead 2 - Device to Host : %ldms\n", et - st);
	
	//check results
	bool result = true;
	for (int i = 0; i < NUM_DATA; i++){
		if(a[i] + b[i] != c[i]){
			printf("[%d] The results is not matched! (%d, %d)\n", i, a[i] + b[i], c[i]);
			result = false;
		}
	}

	if(result)
		printf("GPU works well!\n");

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	delete[] a;
	delete[] b;
	delete[] c;

	return 0;
}

