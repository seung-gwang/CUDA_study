#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>

#define NUM_DATA 65536 //
#define GRID_NUM 64// Make grid that has dimension of (GRID_NUM,1,1)
__global__ void vecAdd(int* _a, int* _b, int* _c){
	//1)find the block 2)find the thread within the block
	int tID =(blockIdx.x * blockDim.x) +  threadIdx.x;
	_c[tID] = _a[tID] + _b[tID];
}

int main(void){
	int *a, *b, *c, *host_c;
	int *d_a, *d_b, *d_c;
	
	double hostTime = 0.0;
	double deviceTime = 0.0;

	int memSize = sizeof(int)*NUM_DATA;

	printf("%d elements, memsize = %d bytes\n", NUM_DATA, memSize);
	
	a = new int[NUM_DATA]; memset(a, 0, memSize);
	b = new int[NUM_DATA]; memset(b, 0, memSize);
	c = new int[NUM_DATA]; memset(c, 0, memSize);
	host_c = new int[NUM_DATA]; memset(host_c, 0, memSize);

	for (int i = 0; i < NUM_DATA; i++){
		a[i] = rand() % 10;
		b[i] = rand() % 10;
	}

	hipMalloc(&d_a, memSize);
	hipMalloc(&d_b, memSize);
	hipMalloc(&d_c, memSize);
	
	//Data Transfer overhead 1
	clock_t st = clock();
	hipMemcpy(d_a, a, memSize, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, memSize, hipMemcpyHostToDevice);
	clock_t et = clock();
	deviceTime += (double)(et-st)/CLOCKS_PER_SEC;
	printf("Data Transfer Overhead 1 - Host to Device : %f seconds\n", (double)(et - st)/CLOCKS_PER_SEC);
	
	
	dim3 grid(GRID_NUM);// GRID : (GRID_NUM,1,1)
	dim3 block(NUM_DATA / GRID_NUM);//NUM_DATA / GRID_NUM == "NUMBER OF THREADS" ... should be <= 1024 
	
	//computation on Device
	st = clock();
	//vecAdd<<<1, NUM_DATA>>>(d_a, d_b, d_c);
	vecAdd<<<grid, block>>>(d_a, d_b, d_c);
	et = clock();
	deviceTime += (double)(et-st)/CLOCKS_PER_SEC;	
	printf("Vector Summation Kernel computation : %f seconds\n",(double)(et - st)/CLOCKS_PER_SEC);
	
	//computation on Host
	st = clock();
	for(int i = 0; i < NUM_DATA; i++)
		host_c[i] = a[i] + b[i];
	et = clock();
	hostTime += (double)(et-st)/CLOCKS_PER_SEC;
	printf("Vector Summation Host computation : %f seconds\n", (double)(et - st)/CLOCKS_PER_SEC);



	//Data Transfer overhead 2
	st = clock();
	hipMemcpy(c, d_c, memSize, hipMemcpyDeviceToHost);
	et = clock();
	deviceTime += (double)(et-st)/CLOCKS_PER_SEC;
	printf("Data Transfer Overhead 2 - Device to Host : %f seconds\n", (double)(et - st)/CLOCKS_PER_SEC);
	
	//check results
	printf("**********RESULT**********\n");
	
	printf("grid dimension : (%d,%d,%d)\n", grid.x, grid.y, grid.z);
	printf("block dimension : (%d,%d,%d)\n", block.x, block.y, block.z);
	bool result = true;
	for (int i = 0; i < NUM_DATA; i++){
		if(a[i] + b[i] != c[i]){
			printf("[%d] The results is not matched! (%d, %d)\n", i, a[i] + b[i], c[i]);
			result = false;
		}
	}

	if(result)
		printf("Calculated Correctly! GPU works well!\n");
	printf("Total Device Time: %f\n", deviceTime);
	printf("Total Host Time: %f\n", hostTime);
	if (deviceTime - hostTime < 0.0)
		printf("Using GPU made the calculation faster\n");
	
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	delete[] a;
	delete[] b;
	delete[] c;
	delete[] host_c;

	return 0;
}

