#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>

#define ELEM_NUM 1024
#define BLOCKX 20
__global__ void checkIndex(void){
	printf("threadIdx: (%d, %d, %d) blockIdx: (%d, %d, %d) blockDim: (%d, %d, %d) gridDim: (%d, %d, %d)\n"
			, threadIdx.x, threadIdx.y, threadIdx.z
			, blockIdx.x, blockIdx.y, blockIdx.z
			, blockDim.x, blockDim.y, blockDim.z
			, gridDim.x, gridDim.y, gridDim.z);

}

int main(int argc, char **argv){
	int nElem = ELEM_NUM;
	
	//Blcok의 모양 지정
	dim3 block(BLOCKX);

	//Grid의 모양 지정  
	dim3 grid((nElem + block.x - 1) / block.x);

	printf("grid.x %d grid.y %d grid.z %d\n", grid.x, grid.y, grid.z);
	printf("block.x %d block.y %d block.z %d\n", block.x, block.y, block.z);
	
	printf("----------------------------Result of checkIndex()----------------------------\n");
	checkIndex <<<grid, block>>> ();
	hipDeviceReset();
	return(0);
}

