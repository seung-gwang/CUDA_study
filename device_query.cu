#include "book.h"

int main(void){
	hipDeviceProp_t prop;

	int count;
	HANDLE_ERROR( hipGetDeviceCount( &count));

	for (int i = 0; i < count; i++){
		HANDLE_ERROR( hipGetDeviceProperties( &prop, i));
		printf("Total global mem: %ld\n", prop.totalGlobalMem);
		//other queries can be made here

	}
}
